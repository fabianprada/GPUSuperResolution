#include "hip/hip_runtime.h"
/*
Copyright (c) 2018, Fabian Prada
All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution.

Neither the name of the Johns Hopkins University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.
*/


#include "gpu-solver.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "cutil.h"
#include "cutil_inline_runtime.h"
#include <stdio.h>
#include "file-io.h"

#define BLOCK_SIZE 16
#define PI2 6.2831853

//typedef float2 fcomplex; 
//float * rhs_h = 0;
//float * rhs_d = 0;
//float * symmetric_extended_rhs_d = 0;
//float * symmetric_extended_rhs_h = 0;
//fcomplex * symmetric_extended_rhs_fft_h = 0;
//fcomplex * symmetric_extended_rhs_fft_d = 0;
//unsigned int array_size = 0;

__global__ void ExtendedSymmetricRHS(float *  rhs_d,float *  extended_rhs_d,const unsigned int imgWidth,const unsigned int imgHeight)
{
	unsigned int tx =blockIdx.x*BLOCK_SIZE + threadIdx.x;
	unsigned int ty =blockIdx.y*BLOCK_SIZE + threadIdx.y;
	if(  tx < imgWidth &&  ty < imgHeight ){
		float value = rhs_d[tx + imgWidth*ty];
		extended_rhs_d[tx + 2*imgWidth*ty] = value;
		extended_rhs_d[ 2*imgWidth - 1 - tx + 2*imgWidth*ty] = value;
		extended_rhs_d[tx + 2*imgWidth*(2*imgHeight - 1 - ty)] = value;
		extended_rhs_d[2*imgWidth - 1 - tx + 2*imgWidth*(2*imgHeight - 1 - ty)] = value;
	}
}

__global__ void ExtractRHS(float *  rhs_d,float *  extended_rhs_d,const unsigned int imgWidth,const unsigned int imgHeight)
{
	unsigned int tx =blockIdx.x*BLOCK_SIZE + threadIdx.x;
	unsigned int ty =blockIdx.y*BLOCK_SIZE + threadIdx.y;
	if(  tx < imgWidth &&  ty < imgHeight ){
		rhs_d[tx + imgWidth*ty]=extended_rhs_d[tx + 2*imgWidth*ty];
	}
}

__global__ void SpectralModulation(float2 *  extended_rhs_fft_d,const unsigned int extended_imgWidth,const unsigned int extended_imgHeight, const float finv_imgWidth,const float finv_imgHeight, const float dc)
{
	unsigned int tx =blockIdx.x*BLOCK_SIZE + threadIdx.x;
	unsigned int ty =blockIdx.y*BLOCK_SIZE + threadIdx.y;
	if(  tx < (extended_imgWidth/2 + 1)  &&  ty < extended_imgHeight ){
		if(tx + ty > 0)
		{
			float attenuation_factor = (finv_imgWidth*finv_imgHeight)/ (4.f -2.f*( cos(PI2*(float)tx*finv_imgWidth) + cos(PI2*((float)ty)*finv_imgHeight)));
			extended_rhs_fft_d[tx + ty*(extended_imgWidth/2 + 1) ].x *= attenuation_factor;
			extended_rhs_fft_d[tx + ty*(extended_imgWidth/2 + 1) ].y *= attenuation_factor;
		}
		else
		{
			extended_rhs_fft_d[0].x = 4.f*dc*(finv_imgWidth*finv_imgHeight);
			extended_rhs_fft_d[0].y = 0.f;
		}
	}
}


void GPUSolvers::FFTLaplaceSolver(float *  rhs_d, float * symmetric_extended_rhs_d, float2 * symmetric_extended_rhs_fft_d,  hipfftHandle & fftPlanFwd, hipfftHandle & fftPlanInv, const unsigned int array_width,const unsigned int array_height ,const float dc)
{
	unsigned int extended_array_width = 2*array_width;
	unsigned int extended_array_height = 2*array_height;

	//if(array_size!=array_width*array_height){
	//	array_size = array_width*array_height;
	//	
	//	if(symmetric_extended_rhs_d!=0){
	//		hipFree(symmetric_extended_rhs_d);
	//	}
	//	hipMalloc((void**)&symmetric_extended_rhs_d,extended_array_width*extended_array_height*sizeof(float));

	//	if(symmetric_extended_rhs_fft_d!=0){
	//		hipFree(symmetric_extended_rhs_fft_d);
	//	}
	//	hipMalloc((void**)&symmetric_extended_rhs_fft_d,(extended_array_height )*(extended_array_width/ 2 + 1) *sizeof(fcomplex));
	//}

	unsigned int  blocksW = (unsigned int) ceilf( (float) array_width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) array_height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	//hipEvent_t start;
	//hipEventCreate(&start);
	//hipEvent_t stop;
	//hipEventCreate(&stop);
	//hipEventRecord(start, NULL);

	ExtendedSymmetricRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);

	//hipfftHandle fftPlanFwd, fftPlanInv;
 //   //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
 //   cufftSafeCall( hipfftPlan2d(&fftPlanFwd, extended_array_height, extended_array_width, HIPFFT_R2C) );
 //   cufftSafeCall( hipfftPlan2d(&fftPlanInv, extended_array_height, extended_array_width, HIPFFT_C2R) );

	cufftSafeCall( hipfftExecR2C(fftPlanFwd, (hipfftReal *)symmetric_extended_rhs_d, (hipfftComplex *)symmetric_extended_rhs_fft_d) );
		
	unsigned int  ext_blocksW = (unsigned int) ceilf( (float) (extended_array_width / 2 + 1) / (float) BLOCK_SIZE );
    unsigned int  ext_blocksH = (unsigned int) ceilf( (float) extended_array_height /(float) BLOCK_SIZE );
    dim3 ext_gridDim( ext_blocksW, ext_blocksH, 1 );

	SpectralModulation<<< ext_gridDim, blockDim >>>(symmetric_extended_rhs_fft_d,extended_array_width,extended_array_height, 1.f/(float)extended_array_width,1.f/(float)extended_array_height,dc);

	cufftSafeCall( hipfftExecC2R(fftPlanInv, (hipfftComplex *)symmetric_extended_rhs_fft_d, (hipfftReal *)symmetric_extended_rhs_d) );
	ExtractRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);

	//hipEventRecord(stop, NULL);
	//hipEventSynchronize(stop);
 //   float msecTotal = 0.0f;
 //   hipEventElapsedTime(&msecTotal, start, stop);
	//printf("Time= %.5f msec \n",msecTotal);
}


//void GPUSolvers::FFTLaplaceSolver() // PASSED TEST
//{
//	int array_height = 359;
//	int array_width = 400;
//	const float dc = 57694.f;
//
//	unsigned int extended_array_width = 2*array_width;
//	unsigned int extended_array_height = 2*array_height;
//
//	if(symmetric_extended_rhs_d_size!=array_width*array_height){
//		if(rhs_d!=0){
//			hipFree(rhs_d);
//		}
//		else{
//			hipMalloc((void**)&rhs_d,array_width*array_height*sizeof(float));
//		}
//
//		if(symmetric_extended_rhs_d!=0){
//			hipFree(symmetric_extended_rhs_d);
//		}
//		else{
//			hipMalloc((void**)&symmetric_extended_rhs_d,extended_array_width*extended_array_height*sizeof(float));
//		}
//		if(symmetric_extended_rhs_h!=0){
//			delete symmetric_extended_rhs_h;
//		}
//		else{
//			symmetric_extended_rhs_h = new float[extended_array_width*extended_array_height*sizeof(float)];
//		}
//		if(symmetric_extended_rhs_fft_d!=0){
//			hipFree(symmetric_extended_rhs_fft_d);
//		}
//		else{
//			hipMalloc((void**)&symmetric_extended_rhs_fft_d,(extended_array_height )*(extended_array_width/ 2 + 1) *sizeof(fcomplex));
//		}
//		if(symmetric_extended_rhs_fft_h!=0){
//			delete symmetric_extended_rhs_fft_h;
//		}
//		else{
//			symmetric_extended_rhs_fft_h = new fcomplex[(extended_array_height)*(extended_array_width/ 2 + 1) *sizeof(fcomplex)];
//		}
//		if(rhs_h!=0){
//			delete rhs_h;
//		}
//		else{
//			rhs_h = new float[array_width*array_height*sizeof(float)];
//		}
//	}
//
//	FileIO::readInputdf("input.txt",rhs_h,array_height*array_width);
//
//	printf("input [0] = %g \n",rhs_h[0]);
//	printf("input [1] = %g \n",rhs_h[1]);
//	printf("input [width - 1] = %g \n",rhs_h[array_width-1]);
//	printf("input [width] = %g \n",rhs_h[array_width]);
//
//	hipMemcpy(rhs_d,rhs_h,array_width*array_height*sizeof(float),hipMemcpyHostToDevice);
//
//	unsigned int  blocksW = (unsigned int) ceilf( (float) array_width / (float) BLOCK_SIZE );
//    unsigned int  blocksH = (unsigned int) ceilf( (float) array_height /(float) BLOCK_SIZE );
//    dim3 gridDim( blocksW, blocksH, 1 );
//    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );
//
//	ExtendedSymmetricRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);
//
//	hipMemcpy(symmetric_extended_rhs_h,symmetric_extended_rhs_d,extended_array_width*extended_array_height*sizeof(float),hipMemcpyDeviceToHost);
//
//	printf("extended [0] = %g \n",symmetric_extended_rhs_h[0]);
//	printf("extended [1] = %g \n",symmetric_extended_rhs_h[1]);
//	printf("extended [array_width-2] = %g \n",symmetric_extended_rhs_h[array_width-2]);
//	printf("extended [array_width-1] = %g \n",symmetric_extended_rhs_h[array_width-1]);
//	printf("extended [array_width] = %g \n",symmetric_extended_rhs_h[array_width]);
//	printf("extended [array_width+1] = %g \n",symmetric_extended_rhs_h[array_width+1]);
//	printf("extended [array_width+2] = %g \n",symmetric_extended_rhs_h[array_width+2]);
//	printf("extended [extended_array_width-1] = %g \n",symmetric_extended_rhs_h[extended_array_width-1]);
//	printf("extended [extended_array_width] = %g \n",symmetric_extended_rhs_h[extended_array_width]);
//
//	hipfftHandle fftPlanFwd, fftPlanInv;
//    //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
//    cufftSafeCall( hipfftPlan2d(&fftPlanFwd, extended_array_height, extended_array_width, HIPFFT_R2C) );
//    cufftSafeCall( hipfftPlan2d(&fftPlanInv, extended_array_height, extended_array_width, HIPFFT_C2R) );
//	cufftSafeCall( hipfftExecR2C(fftPlanFwd, (hipfftReal *)symmetric_extended_rhs_d, (hipfftComplex *)symmetric_extended_rhs_fft_d) );
//		
//	hipMemcpy(symmetric_extended_rhs_fft_h,symmetric_extended_rhs_fft_d,(extended_array_height)*(extended_array_width / 2 + 1) *sizeof(fcomplex),hipMemcpyDeviceToHost);
//
//	printf("extended fft [0] = (%g,%g) \n",symmetric_extended_rhs_fft_h[0].x,symmetric_extended_rhs_fft_h[0].y);
//	printf("extended fft [1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[1].x,symmetric_extended_rhs_fft_h[1].y);
//	printf("extended fft [array_width-2] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width-2].x,symmetric_extended_rhs_fft_h[array_width-2].y);
//	printf("extended fft [array_width-1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width-1].x,symmetric_extended_rhs_fft_h[array_width-1].y);
//	printf("extended fft [array_width] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width].x,symmetric_extended_rhs_fft_h[array_width].y);
//	printf("extended fft [array_width+1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width+1].x,symmetric_extended_rhs_fft_h[array_width+1].y);
//	printf("extended fft [array_width+2] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width+2].x,symmetric_extended_rhs_fft_h[array_width+2].y);
//	printf("extended fft [extended_array_width-1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[extended_array_width-1].x,symmetric_extended_rhs_fft_h[extended_array_width-1].y);
//	printf("extended fft [extended_array_width] = (%g,%g) \n",symmetric_extended_rhs_fft_h[extended_array_width].x,symmetric_extended_rhs_fft_h[extended_array_width].y);
//
//
//	unsigned int  ext_blocksW = (unsigned int) ceilf( (float) (extended_array_width / 2 + 1) / (float) BLOCK_SIZE );
//    unsigned int  ext_blocksH = (unsigned int) ceilf( (float) extended_array_height /(float) BLOCK_SIZE );
//    dim3 ext_gridDim( ext_blocksW, ext_blocksH, 1 );
//
//	//hipEvent_t start;
//	//hipEventCreate(&start);
//	//hipEvent_t stop;
//	//hipEventCreate(&stop);
//	//hipEventRecord(start, NULL);
//	
//	SpectralModulation<<< ext_gridDim, blockDim >>>(symmetric_extended_rhs_fft_d,extended_array_width,extended_array_height, 1.f/(float)extended_array_width,1.f/(float)extended_array_height,dc);
//	
//	hipMemcpy(symmetric_extended_rhs_fft_h,symmetric_extended_rhs_fft_d,(extended_array_height)*(extended_array_width / 2 + 1) *sizeof(fcomplex),hipMemcpyDeviceToHost);
//
//	printf("extended modulated fft [0] = (%g,%g) \n",symmetric_extended_rhs_fft_h[0].x,symmetric_extended_rhs_fft_h[0].y);
//	printf("extended modulated fft [1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[1].x,symmetric_extended_rhs_fft_h[1].y);
//	printf("extended modulated fft [array_width-2] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width-2].x,symmetric_extended_rhs_fft_h[array_width-2].y);
//	printf("extended modulated fft [array_width-1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width-1].x,symmetric_extended_rhs_fft_h[array_width-1].y);
//	printf("extended modulated fft [array_width] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width].x,symmetric_extended_rhs_fft_h[array_width].y);
//	printf("extended modulated fft [array_width+1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width+1].x,symmetric_extended_rhs_fft_h[array_width+1].y);
//	printf("extended modulated fft [array_width+2] = (%g,%g) \n",symmetric_extended_rhs_fft_h[array_width+2].x,symmetric_extended_rhs_fft_h[array_width+2].y);
//	printf("extended modulated fft [extended_array_width-1] = (%g,%g) \n",symmetric_extended_rhs_fft_h[extended_array_width-1].x,symmetric_extended_rhs_fft_h[extended_array_width-1].y);
//	printf("extended modulated fft [extended_array_width] = (%g,%g) \n",symmetric_extended_rhs_fft_h[extended_array_width].x,symmetric_extended_rhs_fft_h[extended_array_width].y);
//
//
//	//hipEventRecord(stop, NULL);
//	//hipEventSynchronize(stop);
// //   float msecTotal = 0.0f;
// //   hipEventElapsedTime(&msecTotal, start, stop);
//	//printf("Time= %.5f msec \n",msecTotal);
//
//
//	cufftSafeCall( hipfftExecC2R(fftPlanInv, (hipfftComplex *)symmetric_extended_rhs_fft_d, (hipfftReal *)symmetric_extended_rhs_d) );
//	hipMemcpy(symmetric_extended_rhs_h,symmetric_extended_rhs_d,extended_array_width*extended_array_height*sizeof(float),hipMemcpyDeviceToHost);
//	
//	printf("extended [0] = %g \n",symmetric_extended_rhs_h[0]);
//	printf("extended [1] = %g \n",symmetric_extended_rhs_h[1]);
//	printf("extended [array_width-2] = %g \n",symmetric_extended_rhs_h[array_width-2]);
//	printf("extended [array_width-1] = %g \n",symmetric_extended_rhs_h[array_width-1]);
//	printf("extended [array_width] = %g \n",symmetric_extended_rhs_h[array_width]);
//	printf("extended [array_width+1] = %g \n",symmetric_extended_rhs_h[array_width+1]);
//	printf("extended [array_width+2] = %g \n",symmetric_extended_rhs_h[array_width+2]);
//	printf("extended [extended_array_width-1] = %g \n",symmetric_extended_rhs_h[extended_array_width-1]);
//	printf("extended [extended_array_width] = %g \n",symmetric_extended_rhs_h[extended_array_width]);
//
//	ExtractRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);
//	hipMemcpy(rhs_h,rhs_d,array_width*array_height*sizeof(float),hipMemcpyDeviceToHost);
//
//	printf("res [0] = %g \n",rhs_h[0]);
//	printf("res [1] = %g \n",rhs_h[1]);
//	printf("res [array_width-2] = %g \n",rhs_h[array_width-2]);
//	printf("res [array_width-1] = %g \n",rhs_h[array_width-1]);
//	printf("res [array_width] = %g \n",rhs_h[array_width]);
//	printf("res [array_width+1] = %g \n",rhs_h[array_width+1]);
//	printf("res [array_width+2] = %g \n",rhs_h[array_width+2]);
//
//	FileIO::writeOutputff("output.txt",rhs_h,array_height*array_width);
//}

//void GPUSolvers::FFTLaplaceSolver(float *  rhs_d, const unsigned int array_width,const unsigned int array_height)
//{
//	unsigned int extended_array_width = 2*array_width;
//	unsigned int extended_array_height = 2*array_height;
//
//	if(symmetric_extended_rhs_d_size!=array_width*array_height){
//		if(symmetric_extended_rhs_d!=0){
//			hipFree(symmetric_extended_rhs_d);
//		}
//		else{
//			hipMalloc((void**)&symmetric_extended_rhs_d,extended_array_width*extended_array_height*sizeof(float));
//		}
//		if(symmetric_extended_rhs_fft_d!=0){
//			hipFree(symmetric_extended_rhs_fft_d);
//		}
//		else{
//			hipMalloc((void**)&symmetric_extended_rhs_fft_d,extended_array_height*(extended_array_width / 2 + 1) *sizeof(fcomplex));
//		}
//	}
//
//
//
//	hipMemcpy(rhs_h,rhs_d,array_width*array_height*sizeof(float),hipMemcpyDeviceToHost);
//
//	printf("Entry 0 = %f", rhs_h[0]);
//
//	unsigned int  blocksW = (unsigned int) ceilf( (float) array_width / (float) BLOCK_SIZE );
//    unsigned int  blocksH = (unsigned int) ceilf( (float) array_height /(float) BLOCK_SIZE );
//    dim3 gridDim( blocksW, blocksH, 1 );
//    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );
//
//	ExtendedSymmetricRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);
//
//	hipfftHandle fftPlanFwd, fftPlanInv;
//    //printf("...creating R2C & C2R FFT plans for %i x %i\n", fftH, fftW);
//    cufftSafeCall( hipfftPlan2d(&fftPlanFwd, extended_array_height, extended_array_width, HIPFFT_R2C) );
//    cufftSafeCall( hipfftPlan2d(&fftPlanInv, extended_array_height, extended_array_width, HIPFFT_C2R) );
//	cufftSafeCall( hipfftExecR2C(fftPlanFwd, (hipfftReal *)symmetric_extended_rhs_d, (hipfftComplex *)symmetric_extended_rhs_fft_d) );
//
//	unsigned int  ext_blocksW = (unsigned int) ceilf( (float) (extended_array_width / 2 + 1) / (float) BLOCK_SIZE );
//    unsigned int  ext_blocksH = (unsigned int) ceilf( (float) extended_array_height /(float) BLOCK_SIZE );
//    dim3 ext_gridDim( ext_blocksW, ext_blocksH, 1 );
//
//	//hipEvent_t start;
//	//hipEventCreate(&start);
//	//hipEvent_t stop;
//	//hipEventCreate(&stop);
//	//hipEventRecord(start, NULL);
//	
//	SpectralInversion<<< ext_gridDim, blockDim >>>(symmetric_extended_rhs_fft_d,extended_array_width,extended_array_height, 1.f/(float)extended_array_width,1.f/(float)extended_array_height);
//	
//	//hipEventRecord(stop, NULL);
//	//hipEventSynchronize(stop);
// //   float msecTotal = 0.0f;
// //   hipEventElapsedTime(&msecTotal, start, stop);
//	//printf("Time= %.5f msec \n",msecTotal);
//
//
//	cufftSafeCall( hipfftExecC2R(fftPlanInv, (hipfftComplex *)symmetric_extended_rhs_fft_d, (hipfftReal *)symmetric_extended_rhs_d) );
//
//	hipMemcpy(symmetric_extended_rhs_fft_h,symmetric_extended_rhs_fft_d,extended_array_height*(extended_array_width / 2 + 1) *sizeof(fcomplex),hipMemcpyDeviceToHost);
//
//	ExtractRHS<<< gridDim, blockDim >>>( rhs_d, symmetric_extended_rhs_d, array_width, array_height);
//}

