#include "hip/hip_runtime.h"
/*
Copyright (c) 2018, Fabian Prada
All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution.

Neither the name of the Johns Hopkins University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.
*/


#include "advection-cuda.cuh"

////////////////////////////////////////////////// FILTERING //////////////////////////////////////////////////

#define BSPLINE5 0
#define BSPLINE3 0
#define GAUSSIAN 1

#if GAUSSIAN
#define PI_2 6.28318530718
#define PI_2_SQRT 2.50662827463

#define SIGMA_SQUARED 2
#define SIGMA  2
#define FILTER_RADIUS_I 6
#define FILTER_RADIUS_F 6
__constant__ float GaussianInteger[2*FILTER_RADIUS_I];

#elif BSPLINE3
#define FILTER_RADIUS_I 2
#define FILTER_RADIUS_F 2.0
#elif BSPLINE5
#define FILTER_RADIUS_I 3
#define FILTER_RADIUS_F 3.0
#else
#define FILTER_RADIUS_I 1
#define FILTER_RADIUS_F 1.0
#endif

__device__ float BilinearFilter(const float x)
{
	float r = 1.f - abs(x);
	if(r > 0.f)
		return r;
	else
		return 0.f;
}

__device__ float BilinearFilter(const float x, const float y)
{
   return BilinearFilter(x)*BilinearFilter(y);
}

__device__ float Bspline3(const float x)
{
	float r = abs(x);
	if (r < 1.f) return (4.f + r*r*(-6.f + 3.f*r)) / 6.f;
	else if (r < 2.f) return  (8.f + r*(-12.f + (6.f - r)*r)) / 6.f;
	else return 0.f;
}

__device__ float Bspline3_d(const float x)
{
		float sign_x = x > 0.f ? 1.f : -1.f;
		float r = abs(x);
		if (r < 1.f){
			return (r*(r*3.f - 4.f) / 2.f)*sign_x;
		}
		else if (r < 2.f){
			return ((r*(4.f - r) - 4.f) / 2.f)*sign_x;
		}
		else return 0.f;
}

__device__ float Bspline3_dd(const float x)
{
	float r = abs(x);
	if (r < 1.f){
		return (3.f*r-2.f);
	}
	else if (r < 2.f){
		return 2.f -r;
	}
	else return 0.f;
}

__device__ float Bspline3(const float x, const float y)
{
	return Bspline3(x)*Bspline3(y);
}


__device__ float Bspline5(const float x)
{
	float r = abs(x);
	if (r < 1.f) return (66.f + r*r*(-60.f
		+ (30.f - 10.f*r)*r*r)) / 120.f;
	else if (r < 2.f) return (51.f + r*(75.f + r*(-210.f
		+ r*(150.f + r*(-45.f + 5.f*r))))) / 120.f;
	else if (r < 3.f) return (243.f + r*(-405.f + r*(270.f
		+ r*(-90.f + (15.f - r)*r)))) / 120.f;
	else return 0.f;
}

__device__ float Bspline5_d(const float x)
{
	float sign_r = x > 0.f ? 1.f : -1.f;
	float r = abs(x);
	if (r < 1.f){
		return  (r*(-1.f + (r*r*(1.f - (5.f*r/ 12.f)))))*sign_r;
	}
	else if (r < 2.f){
		return  ((15.f + r*(-84.f + r*(90.f + r*(-36.f + 5.f*r))))/24.f)*sign_r;
	}
	else if (r < 3.f) return (-(r - 3.f)*(r - 3.f)*(r - 3.f)*(r - 3.f)/ 24.f)*sign_r;
	else return 0.f;
}

__device__ float Bspline5_dd(const float x)
{
	float r = abs(x);
	if (r < 1.f){
		return  -1.f + r*r*(3.f  - (5.f*r/ 3.f));
	}
	else if (r < 2.f){
		return  (-21.f + r*(45 + r*(-27.f + 5*r)))/6.f;
	}
	else if (r < 3.f) return  (-(r - 3.f)*(r - 3.f)*(r - 3.f) / 6.f);
	else return 0.f;
}

#if GAUSSIAN
__device__ float Gaussian(const float u ,const float v)
{
    return exp(-(u*u + v*v) / (2.f*SIGMA_SQUARED)) / (PI_2*SIGMA_SQUARED);
}
__device__ float Gaussian(const float u)
{
    return exp(-(u*u) / (2.f*SIGMA_SQUARED)) / (PI_2_SQRT*SIGMA);
}

extern "C"
void SetGaussianInteger()
{
    float values[2*FILTER_RADIUS_I];

    for (int i = -FILTER_RADIUS_I + 1; i < FILTER_RADIUS_I + 1; i++){
       values[i+FILTER_RADIUS_I-1] = exp(-((float)(i*i)) / (2.f*SIGMA_SQUARED));
    }
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(GaussianInteger), values, sizeof(float)*(2*FILTER_RADIUS_I)));
}

#endif


////////////////////////////////////////////////// CASTING //////////////////////////////////////////////////

__device__  void AdditionFloat4 (float4 & in,const float4 a,const float4 b)
{
	in.x = a.x +b.x;
	in.y = a.y +b.y;
	in.z = a.z +b.z;
	in.w = a.w +b.w;
}

__device__  float3 SubstractionFloat4ToFloat3(const float4 a,const float4 b)
{
	return make_float3(a.x - b.x,a.y - b.y, a.z - b.z);
}

__device__  float4 SubstractionFloat4(const float4 a,const float4 b)
{
	return make_float4(a.x - b.x,a.y - b.y, a.z - b.z,a.w - b.w);
}

__device__  float3 ScalarMultiplicationFloat3(const float3 a,const float s)
{
	return make_float3(a.x*s,a.y*s,a.z*s);
}

__device__ void NormalFloat2UChar(const float x, unsigned char & u)
{
	int i = (int)floor(x*255.f);
	if (i < 0)
		i = 0;
	else if (i >255)
		i = 255;

	u = (unsigned char) i;
}

__device__ unsigned char NormalFloat2UChar(const float x)
{
	int i = (int)floor(x*255.f);
	if (i < 0)
		i = 0;
	else if (i >255)
		i = 255;

	return (unsigned char) i;
}



////////////////////////////////////////////////// COMMOM METHODS //////////////////////////////////////////////////

__global__ void SamplePreviousLevel_Kernel(float4 * color_field_buffer,float * scalar_field_buffer,const float corner_w,const float corner_h, const float relative_scale, const int p_width, const int p_height)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < p_width && global_id_y < p_height ){

		float pos_w = corner_w + relative_scale*((float)(global_id_x)+0.5f);
		float pos_h = corner_h + relative_scale*((float)(global_id_y)+0.5f);

		float4 sample =  tex2D(color_texture,pos_w,pos_h);
		int write_pos = global_id_x + global_id_y*p_width;

		color_field_buffer[write_pos] = sample;
		scalar_field_buffer[write_pos] =sample.x*0.3f + sample.y*0.59f + sample.z*0.11f; 
	}
}


__global__ void SampleGlobalAdvectionField_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht,  const float finv_imgHeight, const float corner_w,const float corner_h,const float scale,  bool p_normalize_advection_field )
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;

	float global_pos_w = corner_w + scale*local_pos_w;
	float global_pos_h = corner_h + scale*local_pos_h;
	
	float cw = floor(global_pos_w);
	float ch = floor(global_pos_h);

	float dw = global_pos_w-cw;
	float dh = global_pos_h-ch;

	float filter_pos_w;
	float filter_pos_h;

	float grad_x = 0.f; 
	float grad_y = 0.f;
	float hessian_xx = 0.f;
	float hessian_yy = 0.f;
	float hessian_xy = 0.f;

	float filter_value;
	float scalar_value;

#if BSPLINE3
	float bspline_x;
	float bspline_y;
	float bspline_d_x;
	float bspline_d_y;
#endif

#if BSPLINE5
	float bspline_x;
	float bspline_y;
	float bspline_d_x;
	float bspline_d_y;
#endif

	for (int ih = -FILTER_RADIUS_I + 1; ih < FILTER_RADIUS_I + 1; ih++){
			for (int iw = -FILTER_RADIUS_I + 1; iw < FILTER_RADIUS_I + 1; iw++){
				filter_pos_w = dw - (float)(iw); // CONVOLUTION POSITION
				filter_pos_h = dh - (float)(ih);
				scalar_value = tex2D(scalar_texture,cw + (float)iw,ch + (float)ih);
#if GAUSSIAN
				filter_value =  Gaussian(filter_pos_w,filter_pos_h);
				grad_x +=(-filter_pos_w / SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y +=(-filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
				hessian_xx += ((filter_pos_w*filter_pos_w - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_yy += ((filter_pos_h*filter_pos_h - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_xy += ((filter_pos_w*filter_pos_h)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
#elif BSPLINE3
				bspline_x =Bspline3(filter_pos_w);
				bspline_y =Bspline3(filter_pos_h);
				bspline_d_x = Bspline3_d(filter_pos_w);
				bspline_d_y = Bspline3_d(filter_pos_h);
				grad_x += bspline_d_x*bspline_y*scalar_value;
				grad_y += bspline_d_y*bspline_x*scalar_value;
				hessian_xx += Bspline3_dd(filter_pos_w)*bspline_y*scalar_value;
				hessian_yy += Bspline3_dd(filter_pos_h)*bspline_x*scalar_value;
				hessian_xy += bspline_d_x*bspline_d_y*scalar_value;
#elif BSPLINE5
				bspline_x =Bspline5(filter_pos_w);
				bspline_y =Bspline5(filter_pos_h);
				bspline_d_x = Bspline5_d(filter_pos_w);
				bspline_d_y = Bspline5_d(filter_pos_h);
				grad_x += bspline_d_x*bspline_y*scalar_value;
				grad_y += bspline_d_y*bspline_x*scalar_value;
				hessian_xx += Bspline5_dd(filter_pos_w)*bspline_y*scalar_value;
				hessian_yy += Bspline5_dd(filter_pos_h)*bspline_x*scalar_value;
				hessian_xy += bspline_d_x*bspline_d_y*scalar_value;
#endif
			}
	}
	scalar_value =hessian_xx*grad_x*grad_x + 2.f*hessian_xy*grad_x*grad_y +hessian_yy*grad_y*grad_y;
	grad_x *= scalar_value;
	grad_y *= scalar_value;

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

__global__ void SampleGlobalAdvectionField_Constant_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht,  const float finv_imgHeight, const float corner_w,const float corner_h,const float scale,  bool p_normalize_advection_field )
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float dw = corner_w + scale*((float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f);
	float dh = corner_h + scale*((float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f);
	
	float cw = floor(dw);
	float ch = floor(dh);

	dw -= cw;
	dh -= ch;

	float filter_pos_w;
	float filter_pos_h;

	float exp_2dw = exp(2.f*dw / (2.f*SIGMA_SQUARED));
	float exp_2dh = exp(2.f*dh / (2.f*SIGMA_SQUARED));

	float exp_dw_first = exp(-(dw*(dw+2.f*(float)(FILTER_RADIUS_I-1)))/(2.f*SIGMA_SQUARED))/(PI_2*SIGMA_SQUARED);
	float exp_dh_first = exp(-(dh*(dh+2.f*(float)(FILTER_RADIUS_I-1)))/(2.f*SIGMA_SQUARED));

	float grad_x = 0.f; 
	float grad_y = 0.f;
	float hessian_xx = 0.f;
	float hessian_yy = 0.f;
	float hessian_xy = 0.f;

	float filter_value;
	float temporal_filter_value;
	float scalar_value;

	for (int ih = -FILTER_RADIUS_I + 1; ih < FILTER_RADIUS_I + 1; ih++){
		    temporal_filter_value = exp_dw_first*exp_dh_first*GaussianInteger[ih + FILTER_RADIUS_I - 1];
			filter_pos_h = dh - (float)(ih);
			for (int iw = -FILTER_RADIUS_I + 1; iw < FILTER_RADIUS_I + 1; iw++){

				filter_pos_w = dw - (float)(iw); // CONVOLUTION POSITION
				scalar_value = tex2D(scalar_texture,cw + (float)iw,ch + (float)ih);

				filter_value = temporal_filter_value * GaussianInteger[iw + FILTER_RADIUS_I - 1];
				grad_x +=(-filter_pos_w / SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y +=(-filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
				hessian_xx += ((filter_pos_w*filter_pos_w - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_yy += ((filter_pos_h*filter_pos_h - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_xy += ((filter_pos_w*filter_pos_h)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;

				temporal_filter_value*=exp_2dw;
			}
		    exp_dh_first*=exp_2dh;
	}

	scalar_value =hessian_xx*grad_x*grad_x + 2.f*hessian_xy*grad_x*grad_y +hessian_yy*grad_y*grad_y;
	grad_x *= scalar_value;
	grad_y *= scalar_value;

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}


__global__ void SampleGlobalMeanGradientField_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht,  const float finv_imgHeight, const float corner_w,const float corner_h,const float scale,  bool p_normalize_advection_field )
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;

	float global_pos_w = corner_w + scale*local_pos_w;
	float global_pos_h = corner_h + scale*local_pos_h;
	
	float cw = floor(global_pos_w);
	float ch = floor(global_pos_h);

	float dw = global_pos_w-cw;
	float dh = global_pos_h-ch;

	float filter_pos_w;
	float filter_pos_h;

	float grad_x = 0.f; 
	float grad_y = 0.f;

	float filter_value;
	float scalar_value;


	for (int iw = -FILTER_RADIUS_I + 1; iw < FILTER_RADIUS_I + 1; iw++){
			for (int ih = -FILTER_RADIUS_I + 1; ih < FILTER_RADIUS_I + 1; ih++){
				filter_pos_w = dw - (float)(iw); // CONVOLUTION POSITION
				filter_pos_h = dh - (float)(ih);
				scalar_value = tex2D(scalar_texture,cw + (float)iw,ch + (float)ih);
#if GAUSSIAN
				filter_value =  Gaussian(filter_pos_w,filter_pos_h);
				grad_x +=(-filter_pos_w / SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y +=(-filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
#elif BSPLINE3
				grad_x += Bspline3_d(filter_pos_w)*Bspline3(filter_pos_h)*scalar_value;
				grad_y += Bspline3_d(filter_pos_h)*Bspline3(filter_pos_w)*scalar_value;
#elif BSPLINE5
				grad_x += Bspline5_d(filter_pos_w)*Bspline5(filter_pos_h)*scalar_value;
				grad_y += Bspline5_d(filter_pos_h)*Bspline5(filter_pos_w)*scalar_value;
#else
#endif
			}
	}


	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

__global__ void SampleLocalAdvectionField_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, bool p_normalize_advection_field)
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;
	
	float filter_pos_w;
	float filter_pos_h;

	float grad_x = 0.f; 
	float grad_y = 0.f;
	float hessian_xx = 0.f;
	float hessian_yy = 0.f;
	float hessian_xy = 0.f;

	float filter_value;
	float scalar_value;

	for (int ih = -FILTER_RADIUS_I + 1; ih < FILTER_RADIUS_I + 1; ih++){
			for (int iw = -FILTER_RADIUS_I + 1; iw < FILTER_RADIUS_I + 1; iw++){
				filter_pos_w = -(float)(iw); // CONVOLUTION POSITION
				filter_pos_h = -(float)(ih);
				scalar_value = tex2D(scalar_texture,local_pos_w - filter_pos_w,local_pos_h - filter_pos_h);
#if GAUSSIAN
				filter_value =  Gaussian(filter_pos_w,filter_pos_h);
				grad_x -=(filter_pos_w / SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y -=(filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
				hessian_xx += ((filter_pos_w*filter_pos_w - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_yy += ((filter_pos_h*filter_pos_h - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_xy += ((filter_pos_w*filter_pos_h)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
#elif BSPLINE3
#else
#endif
			}
	}
	scalar_value =hessian_xx*grad_x*grad_x + 2.f*hessian_xy*grad_x*grad_y +hessian_yy*grad_y*grad_y;
	grad_x *= scalar_value;
	grad_y *= scalar_value;

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

__global__ void SampleLocalMeanGradientField_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, bool p_normalize_advection_field)
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;
	
	float filter_pos_w;
	float filter_pos_h;

	float grad_x = 0.f; 
	float grad_y = 0.f;
	float filter_value;
	float scalar_value;

	for (int ih = -FILTER_RADIUS_I + 1; ih < FILTER_RADIUS_I + 1; ih++){
			for (int iw = -FILTER_RADIUS_I + 1; iw < FILTER_RADIUS_I + 1; iw++){
				filter_pos_w = -(float)(iw); // CONVOLUTION POSITION
				filter_pos_h = -(float)(ih);
				scalar_value = tex2D(scalar_texture,local_pos_w - filter_pos_w,local_pos_h -filter_pos_h);
#if GAUSSIAN
				filter_value =  Gaussian(filter_pos_w,filter_pos_h);
				grad_x -=(filter_pos_w/ SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y -=(filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
#elif BSPLINE3
				grad_x +=Bspline3_d(filter_pos_w)*Bspline3(filter_pos_h)*scalar_value;
				grad_y +=Bspline3_d(filter_pos_h)*Bspline3(filter_pos_w)*scalar_value;
#elif BSPLINE5
				grad_x += Bspline5_d(filter_pos_w)*Bspline5(filter_pos_h)*scalar_value;
				grad_y += Bspline5_d(filter_pos_h)*Bspline5(filter_pos_w)*scalar_value;
#else
#endif
			}
	}

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

__global__ void SampleLocalMeanGradientFieldLinear_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, bool p_normalize_advection_field)
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;
	
	float grad_x = tex2D(scalar_texture,local_pos_w + 1.f,local_pos_h) - tex2D(scalar_texture,local_pos_w - 1.f,local_pos_h); 
	float grad_y = tex2D(scalar_texture,local_pos_w ,local_pos_h + 1.f) - tex2D(scalar_texture,local_pos_w,local_pos_h - 1.f); 

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

__global__ void SetTexture_Kernel( unsigned char * uchar4_buffer_d, float * buffer_r_d,float * buffer_g_d,float * buffer_b_d, const unsigned int imgWidth,const unsigned int imgHeight)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < imgWidth && global_id_y < imgHeight ){

	unsigned int index = global_id_x + global_id_y*imgWidth;
	unsigned int writting_index = 4*index;
	uchar4_buffer_d[writting_index] = NormalFloat2UChar(buffer_r_d[index]);
	uchar4_buffer_d[writting_index + 1] = NormalFloat2UChar(buffer_g_d[index]);
	uchar4_buffer_d[writting_index + 2] = NormalFloat2UChar(buffer_b_d[index]);
	uchar4_buffer_d[writting_index + 3] = 255;
	}

	//if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){
	//	float pos_w = corner_w + scale*((float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f)*finv_imgWidht;
	//	float pos_h = corner_h + scale*((float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f)*finv_imgHeight;
	//	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = tex2D(scalar_texture,pos_w,pos_h);
	//}
}


__global__ void FloatToUchar4( float * src, unsigned char * dst,const unsigned int imgWidth, const unsigned int imgHeight)
{
	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){
	
	int position =(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth; 
	int i_color = (int)floor(src[position]*255.f);
	if (i_color < 0)
		i_color = 0;
	else if (i_color>255)
		i_color = 255;

    unsigned char value = (unsigned char)(i_color);
	dst[4*position] = value;
	dst[4*position + 1] = value;
	dst[4*position + 2] = value;
	dst[4*position + 3] = 255;
	}
}

__global__ void BackwardSamplingFromGlobal_Kernel(float4 * dst, const unsigned int imgWidth, const unsigned int imgHeight, const float corner_w,const float corner_h,const float scale ,const unsigned int num_iter, const unsigned int supersampling, const float angle_tolerance , const float step_amplification)
{
	//int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	//int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;

	float4 cummulative_color = make_float4(0.f,0.f,0.f,0.f);
	float4 sampled_color;

	float cummulative_weight = 0.f;
	float weight;

	float pos_w;
	float pos_h;

	float dw;
	float dh;

	float2 advection_direction;
	float2 last_advection_direction;

	float angle2;
	float advection_direction_norm2;
	float last_advection_direction_norm2;

	for( int iw =0; iw<supersampling; iw++){
		for( int jw =0; jw<supersampling; jw++){

	//dw = - FILTER_RADIUS_F + 2.f*FILTER_RADIUS_F*((float)iw + 0.5f)/((float)supersampling);
	//dh = - FILTER_RADIUS_F + 2.f*FILTER_RADIUS_F*((float)jw + 0.5f)/((float)supersampling);

    dw = -1.f + 2.f*((float)iw + 0.5f)/((float)supersampling);
	dh = -1.f + 2.f*((float)jw + 0.5f)/((float)supersampling);

	pos_w = local_pos_w + dw;
	pos_h = local_pos_h + dh;
	
	advection_direction = tex2D(advection_texture,pos_w,pos_h);
	advection_direction_norm2 = advection_direction.x*advection_direction.x + advection_direction.y*advection_direction.y;

	angle2 = 1.f;

	for(int i =0; i<num_iter; i++){
		if(angle2>angle_tolerance){
			float potential_pos_w = pos_w - advection_direction.x*step_amplification;
			float potential_pos_h = pos_h - advection_direction.y*step_amplification;
			
			if((potential_pos_w > 0.f && potential_pos_w < (float)(imgWidth-1)) && (potential_pos_h > 0.f && potential_pos_h < (float)(imgHeight-1))){
				
			pos_w = potential_pos_w;
			pos_h = potential_pos_h;

			last_advection_direction = advection_direction;
			last_advection_direction_norm2 = advection_direction_norm2;

			advection_direction = tex2D(advection_texture,pos_w,pos_h);
			advection_direction_norm2 = advection_direction.x*advection_direction.x + advection_direction.y*advection_direction.y;

			if(advection_direction_norm2>0.f){
					angle2 = last_advection_direction.x*advection_direction.x + last_advection_direction.y*advection_direction.y;
					angle2 = angle2>0.f ? angle2*angle2 : -angle2*angle2;
					angle2 /=(last_advection_direction_norm2*advection_direction_norm2);
				}
				else{
					angle2 = angle_tolerance - 1.f;
				}
		    }
	   }
	}

	//pos_w = corner_w + scale*(pos_w+0.5f)*finv_imgWidht;
	//pos_h = corner_h + scale*(pos_h+0.5f)*finv_imgHeight;
	pos_w = corner_w + scale*(pos_w); // Unnormalized color texture
	pos_h = corner_h + scale*(pos_h);
	sampled_color = tex2D(color_texture,pos_w,pos_h);

//#if GAUSSIAN
//	weight = Gaussian(dw,dh);
//#elif BSPLINE3
//	weight = Bspline3(dw,dh);
//#else
//	weight = BilinearFilter(dw,dh);
//#endif

	weight = BilinearFilter(dw,dh);

	cummulative_weight += weight; 
	cummulative_color.x += sampled_color.x*weight;
	cummulative_color.y += sampled_color.y*weight;
	cummulative_color.z += sampled_color.z*weight;

		}
	}

	cummulative_color.x/=cummulative_weight;
	cummulative_color.y/=cummulative_weight;
	cummulative_color.z/=cummulative_weight;

	int position =(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth; 

	dst[position] = cummulative_color;
	}
}

__global__ void BackwardSamplingFromLocal_Kernel(float4 * dst, const unsigned int imgWidth, const unsigned int imgHeight,const unsigned int num_iter, const unsigned int supersampling , const float angle_tolerance,const float step_amplification)
{

	if( (blockIdx.x*BLOCK_SIZE + threadIdx.x) < imgWidth && (blockIdx.y*BLOCK_SIZE + threadIdx.y) < imgHeight ){

	float local_pos_w = (float)(blockIdx.x*BLOCK_SIZE + threadIdx.x) + 0.5f;
	float local_pos_h = (float)(blockIdx.y*BLOCK_SIZE + threadIdx.y) + 0.5f;

	float4 cummulative_color = make_float4(0.f,0.f,0.f,0.f);
	float4 sampled_color;

	float cummulative_weight = 0.f;
	float weight;

	float pos_w;
	float pos_h;

	float dw;
	float dh;

	float2 advection_direction;
	float2 last_advection_direction;

	float angle2;
	float advection_direction_norm2;
	float last_advection_direction_norm2;

	for( int iw =0; iw<supersampling; iw++){
		for( int jw =0; jw<supersampling; jw++){

	//dw = - FILTER_RADIUS_F + 2.f*FILTER_RADIUS_F*((float)iw + 0.5f)/((float)supersampling);
	//dh = - FILTER_RADIUS_F + 2.f*FILTER_RADIUS_F*((float)jw + 0.5f)/((float)supersampling);

    dw = -1.f + 2.f*((float)iw + 0.5f)/((float)supersampling);
	dh = -1.f + 2.f*((float)jw + 0.5f)/((float)supersampling);

	pos_w = local_pos_w + dw;
	pos_h = local_pos_h + dh;
	
	advection_direction = tex2D(advection_texture,pos_w,pos_h);
	advection_direction_norm2 = advection_direction.x*advection_direction.x + advection_direction.y*advection_direction.y;

	angle2 = 1.f;

	for(int i=0; i<num_iter; i++){		
		if(angle2>angle_tolerance){
			float potential_pos_w = pos_w - advection_direction.x*step_amplification;
			float potential_pos_h = pos_h - advection_direction.y*step_amplification;
			if((potential_pos_w > 0.f && potential_pos_w < (float)(imgWidth-1)) && (potential_pos_h > 0.f && potential_pos_h < (float)(imgHeight-1))){
				
				pos_w = potential_pos_w;
				pos_h = potential_pos_h;

				last_advection_direction = advection_direction;
				last_advection_direction_norm2 = advection_direction_norm2;

				advection_direction = tex2D(advection_texture,pos_w,pos_h);
				advection_direction_norm2 = advection_direction.x*advection_direction.x + advection_direction.y*advection_direction.y;
				if(advection_direction_norm2>0.f){
					angle2 = last_advection_direction.x*advection_direction.x + last_advection_direction.y*advection_direction.y;
					angle2 = angle2>0.f ? angle2*angle2 : -angle2*angle2;
					angle2 /=(last_advection_direction_norm2*advection_direction_norm2);
				}
				else{
					angle2 = angle_tolerance - 1.f;
				}
			}
		}
	}

	sampled_color = tex2D(color_texture,pos_w,pos_h);

//#if GAUSSIAN
//	weight = Gaussian(dw,dh);
//#elif BSPLINE3
//	weight = Bspline3(dw,dh);
//#else
//	weight = BilinearFilter(dw,dh);
//#endif
	weight = BilinearFilter(dw,dh);

	cummulative_weight += weight; 
	cummulative_color.x += sampled_color.x*weight;
	cummulative_color.y += sampled_color.y*weight;
	cummulative_color.z += sampled_color.z*weight;

		}
	}

	cummulative_color.x/=cummulative_weight;
	cummulative_color.y/=cummulative_weight;
	cummulative_color.z/=cummulative_weight;

	int position =(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth; 

	dst[position] = cummulative_color;
	}
}

///////////////////////////////////////// VISUALIZATION TOOLS ///////////////////////////////////////// 

#include "gltools.h"

__global__ void SetVectorField_Kernel( float * dst, float2 * vector_field, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht, const float finv_imgHeight, bool normalize_for_visualization, bool clamp_for_visualization, const float clamping_threshold, const float visual_amplification)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < imgWidth && global_id_y < imgHeight ){

	float unnormalized_pos_w = ((float)(global_id_x) + 0.5f);
	float unnormalized_pos_h = ((float)(global_id_y) + 0.5f);

	unsigned int reading_index = global_id_x + global_id_y*imgWidth;
	unsigned int writting_index = 9*reading_index;
	
	float2 vector_direction = vector_field[reading_index];


	float vector_direction_norm = vector_direction.x*vector_direction.x + vector_direction.y*vector_direction.y;
	if(vector_direction_norm > 0.f){
		vector_direction_norm = sqrt(vector_direction_norm);
		if(normalize_for_visualization){
			vector_direction.x*= (visual_amplification/vector_direction_norm);
			vector_direction.y*= (visual_amplification/vector_direction_norm);
		}
		else if(clamp_for_visualization && vector_direction_norm > clamping_threshold){
			vector_direction.x*= (visual_amplification*clamping_threshold/vector_direction_norm);
			vector_direction.y*= (visual_amplification*clamping_threshold/vector_direction_norm);
		}
		else{
			vector_direction.x*= visual_amplification;
			vector_direction.y*= visual_amplification;
		}
	}

	dst[writting_index] = (unnormalized_pos_w + vector_direction.x)*finv_imgWidht;
	dst[writting_index + 1] = (unnormalized_pos_h + vector_direction.y)*finv_imgHeight;
	dst[writting_index + 2] = 0.f;
	dst[writting_index + 3] = (unnormalized_pos_w - vector_direction.y * 0.25f)*finv_imgWidht;
	dst[writting_index + 4] = (unnormalized_pos_h + vector_direction.x * 0.25f)*finv_imgHeight;
	dst[writting_index + 5] = 0.f;
	dst[writting_index + 6] = (unnormalized_pos_w + vector_direction.y * 0.25f)*finv_imgWidht;
	dst[writting_index + 7] = (unnormalized_pos_h - vector_direction.x * 0.25f)*finv_imgHeight;
	dst[writting_index + 8] = 0.f;
	}
}

void GLTools::TransformFloat2ArrayToGLVectorBuffer_CUDA(hipGraphicsResource_t& cuda_resource, float2 * vector_buffer, const unsigned int imgWidth,const unsigned int imgHeigh, bool normalize_vector_visualization,bool clamp_for_visualization,const float clamping_threshold,const float field_amplification)
{
	cutilSafeCall( hipGraphicsMapResources( 1 , &cuda_resource ) );
	float * vertex_buffer_handle;
	size_t num_bytes;
	cutilSafeCall( hipGraphicsResourceGetMappedPointer((void **)&vertex_buffer_handle,&num_bytes,cuda_resource));

	unsigned int  blocksW = (unsigned int) ceilf( (float) imgWidth / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) imgHeigh /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	SetVectorField_Kernel<<< gridDim, blockDim >>>( vertex_buffer_handle, vector_buffer, imgWidth, imgHeigh, 1.f/(float)imgWidth, 1.f/(float)imgHeigh,normalize_vector_visualization,clamp_for_visualization,clamping_threshold,field_amplification);

	cutilSafeCall( hipGraphicsUnmapResources( 1, &cuda_resource) );

}

__global__ void SetVectorField_Kernel2( float * dst, float * vector0_field, float * vector1_field, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht, const float finv_imgHeight, bool normalize_for_visualization, bool clamp_for_visualization, const float clamping_threshold, const float visual_amplification)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < imgWidth && global_id_y < imgHeight ){

	float unnormalized_pos_w = ((float)(global_id_x) + 0.5f);
	float unnormalized_pos_h = ((float)(global_id_y) + 0.5f);

	unsigned int reading_index = global_id_x + global_id_y*imgWidth;
	unsigned int writting_index = 9*reading_index;
	
	float2 vector_direction;
	vector_direction.x = vector0_field[reading_index];
	vector_direction.y = vector1_field[reading_index];

	float vector_direction_norm = vector_direction.x*vector_direction.x + vector_direction.y*vector_direction.y;
	if(vector_direction_norm > 0.f){
		vector_direction_norm = sqrt(vector_direction_norm);
		if(normalize_for_visualization){
			vector_direction.x*= (visual_amplification/vector_direction_norm);
			vector_direction.y*= (visual_amplification/vector_direction_norm);
		}
		else if(clamp_for_visualization && vector_direction_norm > clamping_threshold){
			vector_direction.x*= (visual_amplification*clamping_threshold/vector_direction_norm);
			vector_direction.y*= (visual_amplification*clamping_threshold/vector_direction_norm);
		}
		else{
			vector_direction.x*= visual_amplification;
			vector_direction.y*= visual_amplification;
		}
	}

	dst[writting_index] = (unnormalized_pos_w + vector_direction.x)*finv_imgWidht;
	dst[writting_index + 1] = (unnormalized_pos_h + vector_direction.y)*finv_imgHeight;
	dst[writting_index + 2] = 0.f;
	dst[writting_index + 3] = (unnormalized_pos_w - vector_direction.y * 0.25f)*finv_imgWidht;
	dst[writting_index + 4] = (unnormalized_pos_h + vector_direction.x * 0.25f)*finv_imgHeight;
	dst[writting_index + 5] = 0.f;
	dst[writting_index + 6] = (unnormalized_pos_w + vector_direction.y * 0.25f)*finv_imgWidht;
	dst[writting_index + 7] = (unnormalized_pos_h - vector_direction.x * 0.25f)*finv_imgHeight;
	dst[writting_index + 8] = 0.f;
	}
}

void GLTools::Transform2FloatArrayToGLVectorBuffer_CUDA(hipGraphicsResource_t& cuda_resource, float * vector0_buffer,float * vector1_buffer, const unsigned int imgWidth,const unsigned int imgHeigh, bool normalize_vector_visualization,bool clamp_for_visualization,const float clamping_threshold,const float field_amplification)
{
	cutilSafeCall( hipGraphicsMapResources( 1 , &cuda_resource ) );
	float * vertex_buffer_handle;
	size_t num_bytes;
	cutilSafeCall( hipGraphicsResourceGetMappedPointer((void **)&vertex_buffer_handle,&num_bytes,cuda_resource));

	unsigned int  blocksW = (unsigned int) ceilf( (float) imgWidth / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) imgHeigh /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	SetVectorField_Kernel2<<< gridDim, blockDim >>>( vertex_buffer_handle, vector0_buffer,vector1_buffer,imgWidth, imgHeigh, 1.f/(float)imgWidth, 1.f/(float)imgHeigh,normalize_vector_visualization,clamp_for_visualization,clamping_threshold,field_amplification);

	cutilSafeCall( hipGraphicsUnmapResources( 1, &cuda_resource) );
}

__global__ void SetTexture_Kernel(unsigned char * uchar4_buffer_d,float4 * float4_buffer_d,const int p_width, const int p_height){

	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < p_width && global_id_y < p_height ){

	unsigned int index = global_id_x + global_id_y*p_width;
	unsigned int writting_index = 4*index;
	uchar4_buffer_d[writting_index] = NormalFloat2UChar(float4_buffer_d[index].x);
	uchar4_buffer_d[writting_index + 1] = NormalFloat2UChar(float4_buffer_d[index].y);
	uchar4_buffer_d[writting_index + 2] = NormalFloat2UChar(float4_buffer_d[index].z);
	uchar4_buffer_d[writting_index + 3] = 255;
	}
}


void GLTools::TransformFloat4ArrayToGLColorBuffer_CUDA(hipGraphicsResource_t& cuda_resource, float4 * float4_buffer, unsigned char * uchar4_buffer, const unsigned int width,const unsigned int height)
{
	cutilSafeCall( hipGraphicsMapResources( 1, &cuda_resource ) );
	hipArray* texture_buffer_handle;
	cutilSafeCall( hipGraphicsSubResourceGetMappedArray( &texture_buffer_handle, cuda_resource, 0, 0 ) );

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	SetTexture_Kernel<<< gridDim, blockDim >>>(uchar4_buffer,float4_buffer, width, height);

	cutilSafeCall( hipMemcpyToArray( texture_buffer_handle, 0, 0, uchar4_buffer, width*height*4*sizeof(unsigned char), hipMemcpyDeviceToDevice ) );
	cutilSafeCall( hipGraphicsUnmapResources( 1, &cuda_resource) );
}

__global__ void CopyFloatFromFloat4_Kernel(float4 *src, float * dst, const unsigned int width,const unsigned int height, int channel)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < width && global_id_y < height ){
		unsigned int index = global_id_x + global_id_y*width;
		if(channel ==0)
			dst[index] = src[index].x;
		else if(channel ==1)
			dst[index] = src[index].y;
		else if(channel ==2)
			dst[index] = src[index].z;
	}
}

void CopyFloatFromFloat4(float4 *src, float * dst, const unsigned int width,const unsigned int height,const unsigned int channel)
{
	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	CopyFloatFromFloat4_Kernel<<< gridDim, blockDim >>>(src,dst, width, height,channel);
}

__global__ void CopyFloatToFloat4_Kernel(float *src, float4 * dst, const unsigned int width,const unsigned int height, int channel)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < width && global_id_y < height ){
		unsigned int index = global_id_x + global_id_y*width;
		if(channel ==0)
			dst[index].x = src[index];
		else if(channel ==1)
			dst[index].y = src[index];
		else if(channel ==2)
			dst[index].z = src[index];
	}
}

void CopyFloatToFloat4(float *src, float4 * dst, const unsigned int width,const unsigned int height,const unsigned int channel)
{
	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	CopyFloatToFloat4_Kernel<<< gridDim, blockDim >>>(src,dst, width, height,channel);
}

struct sax_functor {
	const float a;
	sax_functor(float p_a) : a(p_a) {}
	__host__ __device__ float operator()(const float& x) const {
		return a * x; } 
};

__global__ void ScaleVectors_Kernel(float2 * vector_buffer, float * norm_buffer, const float scale_value, int width, int height){
	
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < width &&  global_id_y < height ){
		int write_pos = global_id_x + global_id_y*width;
		vector_buffer[write_pos].x*=scale_value;
		vector_buffer[write_pos].y*=scale_value;
		norm_buffer[write_pos]*=scale_value;
	}
}

__global__ void GeneralScaleVectors_Kernel(float2 * vector_buffer, float * norm_buffer, int width, int height){
	
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( global_id_x < width &&  global_id_y < height ){
		int write_pos = global_id_x + global_id_y*width;
		float2 vector_value = vector_buffer[write_pos];
		float initial_norm = vector_value.x*vector_value.x + vector_value.y*vector_value.y;
		if(initial_norm > 0.f){
			initial_norm = sqrt(initial_norm);
			float new_norm = norm_buffer[write_pos];
			new_norm/=initial_norm;
			vector_buffer[write_pos].x*=new_norm;
			vector_buffer[write_pos].y*=new_norm;
		}
	}
}

void GLTools::GeneralVectorScaling(float2 * vector_buffer, float * norm_buffer, int width, int height){
	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	GeneralScaleVectors_Kernel<<< gridDim, blockDim >>>(vector_buffer,norm_buffer,width,height);
}

void GLTools::NormalizeVectorFieldByMaxima(float2 * vector_buffer, float * norm_buffer, int width, int height){

	thrust::device_ptr<float> dev_ptr_norms(norm_buffer);
	float max_value = thrust::reduce(dev_ptr_norms, dev_ptr_norms + height*width, (float) -FLT_MAX, thrust::maximum<float>());

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	ScaleVectors_Kernel<<< gridDim, blockDim >>>(vector_buffer,norm_buffer,1.f/max_value,width,height);
}

///////////////////////////////////////// CUDA ADVECTION OBJECT ///////////////////////////////////////// 
#include "array-advection.h"

void CUDA_Advection_Object::SampleFieldFromGlobal_CUDA( VectorFieldMode vector_mode,float pos_w, float pos_h, bool normalized_coordinates, const float scale, NormalizationMode normalization_mode)
{
	scalar_texture.addressMode[0] = hipAddressModeMirror;
	scalar_texture.addressMode[1] = hipAddressModeMirror;
	scalar_texture.filterMode = hipFilterModeLinear;
	scalar_texture.normalized = false;

	if(normalized_coordinates){
		pos_w *= (float)(width);
		pos_h *= (float)(height);
	}

	cutilSafeCall( hipBindTextureToArray(scalar_texture,global_scalar_field_array) );

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	bool normalize_field = normalization_mode == UNIFORM_NORMALIZATION ? true : false;

	if(vector_mode == MEAN_GRADIENT_FIELD){
		SampleGlobalMeanGradientField_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field);
	}
	else if(vector_mode == ADVECTION_VECTOR_FIELD){
		SampleGlobalAdvectionField_Kernel<<< gridDim, blockDim >>>( float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field);
	}

	if(normalization_mode == MAXIMA_NORMALIZATION){
		GLTools::NormalizeVectorFieldByMaxima(float2_buffer,float_buffer_0,width,height);
	}

    cutilSafeCall( hipUnbindTexture( scalar_texture ) );
}

void CUDA_Advection_Object::SampleFieldFromLocal_CUDA( VectorFieldMode vector_mode, NormalizationMode normalization_mode)
{
	scalar_texture.addressMode[0] = hipAddressModeMirror;
	scalar_texture.addressMode[1] = hipAddressModeMirror;
	scalar_texture.filterMode = hipFilterModeLinear;
	scalar_texture.normalized = false;

	cutilSafeCall( hipBindTextureToArray(scalar_texture,local_scalar_field_array) );

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	bool normalize_field = normalization_mode == UNIFORM_NORMALIZATION ? true : false;

	if(vector_mode == MEAN_GRADIENT_FIELD){
		SampleLocalMeanGradientField_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width,height,normalize_field);
	}
	else if(vector_mode == MEAN_GRADIENT_FIELD_LINEAR){
		SampleLocalMeanGradientFieldLinear_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width,height,normalize_field);
	}
	else if(vector_mode == ADVECTION_VECTOR_FIELD){
		SampleLocalAdvectionField_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width, height,normalize_field);
	}

	if(normalization_mode == MAXIMA_NORMALIZATION){
		GLTools::NormalizeVectorFieldByMaxima(float2_buffer,float_buffer_0,width,height);
	}

    cutilSafeCall( hipUnbindTexture( scalar_texture ) );
}


void CUDA_Advection_Object::SampleField_CUDA(VectorFieldMode p_vector_field_mode, SamplingDomain p_sampling_domain_mode, float pos_w, float pos_h, bool normalized_coordinates, const float scale, NormalizationMode normalization_mode)
{
	if(p_vector_field_mode == MEAN_GRADIENT_FIELD || p_vector_field_mode == ADVECTION_VECTOR_FIELD){
		if(p_sampling_domain_mode == GLOBAL_DOMAIN){
		SampleFieldFromGlobal_CUDA(p_vector_field_mode,pos_w, pos_h, normalized_coordinates, scale, normalization_mode);
		}
		else if(p_sampling_domain_mode == LOCAL_DOMAIN){
		SampleFieldFromLocal_CUDA( p_vector_field_mode,normalization_mode);
		}
		else if(p_sampling_domain_mode == HYBRID_DOMAIN){
		//SampleFieldFromLocal_CUDA( MEAN_GRADIENT_FIELD,normalization_mode);
		SampleFieldFromLocal_CUDA( MEAN_GRADIENT_FIELD_LINEAR,normalization_mode);
		hipMemcpy(float_buffer_1,float_buffer_0,width*height*sizeof(float),hipMemcpyDeviceToDevice);
		//SampleFieldFromGlobal_CUDA(p_vector_field_mode,pos_w, pos_h, normalized_coordinates,scale,NONE_NORMALIZATION);
		//SampleGlobalFieldFromSampledFilter_CUDA(p_vector_field_mode,pos_w, pos_h, normalized_coordinates,scale,NONE_NORMALIZATION);
		SampleGlobalFieldFromConstantFilter_CUDA(p_vector_field_mode,pos_w, pos_h, normalized_coordinates,scale,NONE_NORMALIZATION);
		GLTools::GeneralVectorScaling(float2_buffer,float_buffer_1,width,height);
		}

	}
}

void CUDA_Advection_Object::UpdateAdvectionArray_CUDA(SamplingDomain p_sampling_domain_mode,float pos_w, float pos_h, bool normalized_coordinates, const float scale, NormalizationMode normalization_mode)
{
	SampleField_CUDA(ADVECTION_VECTOR_FIELD,p_sampling_domain_mode,pos_w, pos_h, normalized_coordinates, scale, normalization_mode);
	cutilSafeCall( hipMemcpyToArray(advection_field_array, 0, 0, float2_buffer, width*height*sizeof(float2), hipMemcpyDeviceToDevice ) );
}

void CUDA_Advection_Object::BackwardSampling_CUDA(SamplingDomain color_sampling_domain, hipArray * p_color_field_array, hipArray * p_advection_field_array, const unsigned int num_iter, const unsigned int supersampling, float pos_w, float pos_h, bool normalized_coordinates, const float scale, const float step_amplification)
{
	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	color_texture.addressMode[0] = hipAddressModeMirror;
	color_texture.addressMode[1] = hipAddressModeMirror;
	color_texture.filterMode = hipFilterModeLinear;
	color_texture.normalized = false;

	cutilSafeCall( hipBindTextureToArray(color_texture,p_color_field_array) );

	advection_texture.addressMode[0] = hipAddressModeMirror;
	advection_texture.addressMode[1] = hipAddressModeMirror;
	advection_texture.filterMode = hipFilterModeLinear;
	advection_texture.normalized = false; // NOTE : This may changed if out of range coordinates are used!!

	cutilSafeCall( hipBindTextureToArray(advection_texture,p_advection_field_array) );

	if(normalized_coordinates){
		pos_w *= (float)(width);
		pos_h *= (float)(height);
	}

	//if(level_number==2){
	//	pos_w = 0.0000f;
	//	pos_h = 0.0000f;
	//	scale = 1.f;
	//}

	if(color_sampling_domain == GLOBAL_DOMAIN)
	BackwardSamplingFromGlobal_Kernel<<< gridDim, blockDim >>>(float4_buffer, width, height, pos_w, pos_h, scale, num_iter, supersampling, 0.f,step_amplification);
	else if(color_sampling_domain == LOCAL_DOMAIN)
	BackwardSamplingFromLocal_Kernel<<< gridDim, blockDim >>>(float4_buffer, width, height, num_iter, supersampling, 0.f,step_amplification);

	cutilSafeCall( hipUnbindTexture(color_texture) );
    cutilSafeCall( hipUnbindTexture(advection_texture) );
}


__global__ void SampleGlobalAdvectionFromSampledFilter_Kernel( float2* dst, float * norms, const unsigned int imgWidth,const unsigned int imgHeight, const float finv_imgWidht,  const float finv_imgHeight, const float corner_w,const float corner_h,const float scale,  bool p_normalize_advection_field, float * filter_w, float * filter_h, const int filter_support, const int filter_radius)
{
	int thread_idx = blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int thread_idy = blockIdx.y*BLOCK_SIZE + threadIdx.y;

	if( thread_idx < imgWidth && thread_idy < imgHeight ){

	float dw = corner_w + scale*((float)thread_idx + 0.5f);//global_pos_w
	float dh = corner_h + scale*((float)thread_idy + 0.5f);//global_pos_h
	
	float cw = floor(dw);
	float ch = floor(dh);

	dw-=cw;
	dh-=ch;

	float filter_pos_w;
	float filter_pos_h;

	float grad_x = 0.f; 
	float grad_y = 0.f;
	float hessian_xx = 0.f;
	float hessian_yy = 0.f;
	float hessian_xy = 0.f;

	float filter_value_h;
	float filter_value;
	float scalar_value;

	float shift_h;

	for (int ih = 0; ih < filter_support; ih++){
		filter_value_h = filter_h[imgHeight*ih + thread_idy];//texFetch may be more efficient!!
			shift_h = (float)(ih - filter_radius + 1);
			for(int iw = 0 ; iw < filter_support; iw++){
				//texFetch may be more efficient!!
				filter_pos_w = dw - (float)(iw - filter_radius + 1); // CONVOLUTION POSITION // shift_w
				filter_pos_h = dh - shift_h;
				scalar_value = tex2D(scalar_texture,cw + (float)(iw - filter_radius + 1),ch + shift_h);
				filter_value = filter_value_h*filter_w[imgWidth*iw + thread_idx];//texFetch may be more efficient!!

				grad_x +=(-filter_pos_w / SIGMA_SQUARED)*filter_value*scalar_value;
				grad_y +=(-filter_pos_h / SIGMA_SQUARED)*filter_value*scalar_value;
				hessian_xx += ((filter_pos_w*filter_pos_w - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_yy += ((filter_pos_h*filter_pos_h - SIGMA_SQUARED)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
				hessian_xy += ((filter_pos_w*filter_pos_h)/(SIGMA_SQUARED*SIGMA_SQUARED))*filter_value*scalar_value;
			}
	}

	scalar_value =hessian_xx*grad_x*grad_x + 2.f*hessian_xy*grad_x*grad_y +hessian_yy*grad_y*grad_y;
	grad_x *= scalar_value;
	grad_y *= scalar_value;

	float grad_norm = sqrt(grad_x*grad_x + grad_y*grad_y);

	if(p_normalize_advection_field && grad_norm > 0.f){
			grad_x/=grad_norm;
			grad_y/=grad_norm;
			norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = 1.f;
	}
	else{
		norms[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth] = grad_norm;
	}

	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].x  = grad_x;
	dst[(blockIdx.x*BLOCK_SIZE + threadIdx.x) + (blockIdx.y*BLOCK_SIZE + threadIdx.y)*imgWidth].y  = grad_y;
	}
}

void CUDA_Advection_Object::SampleGlobalFieldFromSampledFilter_CUDA(VectorFieldMode vector_mode,float pos_w, float pos_h, bool normalized_coordinates, const float scale, NormalizationMode normalization_mode)
{
	scalar_texture.addressMode[0] = hipAddressModeMirror;
	scalar_texture.addressMode[1] = hipAddressModeMirror;
	scalar_texture.filterMode = hipFilterModeLinear;
	scalar_texture.normalized = false;

	if(normalized_coordinates){
		pos_w *= (float)(width);
		pos_h *= (float)(height);
	}

	cutilSafeCall( hipBindTextureToArray(scalar_texture,global_scalar_field_array) );

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	bool normalize_field = normalization_mode == UNIFORM_NORMALIZATION ? true : false;

	if(vector_mode == MEAN_GRADIENT_FIELD){
		printf("unimplemented!! \n");		
	}
	else if(vector_mode == ADVECTION_VECTOR_FIELD){
		//SampleGlobalAdvectionField_Kernel<<< gridDim, blockDim >>>( float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field);
	    SampleGlobalAdvectionFromSampledFilter_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field,sampled_filter_values_w,sampled_filter_values_h,filter_support,filter_radius);
	}
	if(normalization_mode == MAXIMA_NORMALIZATION){
		GLTools::NormalizeVectorFieldByMaxima(float2_buffer,float_buffer_0,width,height);
	}
    cutilSafeCall( hipUnbindTexture( scalar_texture ) );
}

void CUDA_Advection_Object::SampleGlobalFieldFromConstantFilter_CUDA(VectorFieldMode vector_mode,float pos_w, float pos_h, bool normalized_coordinates, const float scale, NormalizationMode normalization_mode)
{
	scalar_texture.addressMode[0] = hipAddressModeMirror;
	scalar_texture.addressMode[1] = hipAddressModeMirror;
	scalar_texture.filterMode = hipFilterModeLinear;
	scalar_texture.normalized = false;

	if(normalized_coordinates){
		pos_w *= (float)(width);
		pos_h *= (float)(height);
	}

	cutilSafeCall( hipBindTextureToArray(scalar_texture,global_scalar_field_array) );

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	bool normalize_field = normalization_mode == UNIFORM_NORMALIZATION ? true : false;

	if(vector_mode == MEAN_GRADIENT_FIELD){
		printf("unimplemented!! \n");		
	}
	else if(vector_mode == ADVECTION_VECTOR_FIELD){
		//SampleGlobalAdvectionField_Kernel<<< gridDim, blockDim >>>( float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field);
		SampleGlobalAdvectionField_Constant_Kernel<<< gridDim, blockDim >>>(float2_buffer, float_buffer_0, width, height,  1.f/(float)width, 1.f/(float)height, pos_w, pos_h, scale,normalize_field);
	}
	if(normalization_mode == MAXIMA_NORMALIZATION){
		GLTools::NormalizeVectorFieldByMaxima(float2_buffer,float_buffer_0,width,height);
	}
    cutilSafeCall( hipUnbindTexture( scalar_texture ) );
}

__global__ void FilterSampling_Kernel( float * sampled_values_w,float * sampled_values_h,const int width, const int height, const int  filter_support, const int  filter_radius, const float corner_w, const float corner_h, const float scale)
{
	int thread_id = blockIdx.x*LARGE_BLOCK_SIZE + threadIdx.x;
	float shift;

	if( thread_id < width ){
		shift = corner_w+ scale*((float)thread_id + 0.5f); //global_pos
		shift -= floor(shift);
		shift += (filter_radius - 1.f);
		for(int i=0; i<filter_support; i++){
			sampled_values_w[width*i + thread_id] = Gaussian(shift);
			shift -=1.f;
		}
	}

	if( thread_id < height ){
		shift = corner_h+ scale*((float)thread_id + 0.5f); //global_pos
		shift -= floor(shift);
		shift += (filter_radius - 1.f);
		for(int i=0; i<filter_support; i++){
			sampled_values_h[height*i + thread_id] = Gaussian(shift);
			shift -=1.f;
		}
	}
}

void  CUDA_Advection_Object::UpdateFilterSamples(float pos_w, float pos_h, bool normalized_coordinates, const float scale)
{
	int largest_dimension = width > height ? width : height;
	unsigned int  blocks = (unsigned int) ceilf( (float) largest_dimension / (float) LARGE_BLOCK_SIZE );
    dim3 gridDim( blocks, 1, 1 );
    dim3 blockDim( LARGE_BLOCK_SIZE, 1, 1 );

	if(normalized_coordinates){
		pos_w *= (float)(width);
		pos_h *= (float)(height);
	}

	FilterSampling_Kernel<<< gridDim, blockDim >>>(sampled_filter_values_w,sampled_filter_values_h,width, height, filter_support, filter_radius, pos_w, pos_h, scale);
}


///////////////////////////////////////// ARRAY ADVECTION  ///////////////////////////////////////// 

///////////////////////////////////////// ARRAY HIERARCHY ///////////////////////////////////////// 

#include "array-hierarchy.h"

void Array_Hierarchy::SetTextureToLevel(const int level_number,hipGraphicsResource_t& p_texture_resource)
{
	cutilSafeCall( hipMemcpyFromArray(float4_buffer,hierarchical_color_field_array[level_number],0, 0, width*height*sizeof(float4), hipMemcpyDeviceToDevice ) );
	GLTools::TransformFloat4ArrayToGLColorBuffer_CUDA(p_texture_resource,float4_buffer,uchar4_buffer,width,height);
}

void Array_Hierarchy::SamplePreviousLevel_CUDA(const int level_number){

	// Sample Color and Scalar Field
	color_texture.addressMode[0] = hipAddressModeMirror;
	color_texture.addressMode[1] = hipAddressModeMirror;
	color_texture.filterMode = hipFilterModeLinear;
	color_texture.normalized = false;

	cutilSafeCall( hipBindTextureToArray(color_texture,hierarchical_color_field_array[level_number-1]));

	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	Point<2> relative_position = (level_absolute_coordinate[level_number] - level_absolute_coordinate[level_number-1])/level_absolute_scale[level_number-1];
	float pos_w = (float)relative_position[0]*(float)(width-1);
	float pos_h = (float)relative_position[1]*(float)(height-1);
	float relative_scale = level_absolute_scale[level_number]/level_absolute_scale[level_number-1];

	SamplePreviousLevel_Kernel<<< gridDim, blockDim >>>(float4_buffer,float_buffer_0,pos_w,pos_h,relative_scale,width,height);

    cutilSafeCall( hipUnbindTexture( color_texture ) );
}

void Array_Hierarchy::UpdateLevel(const int num_iter, const int level_number)
{
	SamplePreviousLevel_CUDA(level_number);
	cutilSafeCall( hipMemcpyToArray(local_color_field_array, 0, 0, float4_buffer, width*height*sizeof(float4), hipMemcpyDeviceToDevice ) );
	cutilSafeCall( hipMemcpyToArray(local_scalar_field_array, 0, 0, float_buffer_0, width*height*sizeof(float), hipMemcpyDeviceToDevice ) );
	
	UpdateAdvectionArray_CUDA(HYBRID_DOMAIN,(float)level_absolute_coordinate[level_number][0],(float)level_absolute_coordinate[level_number][1],true,(float)level_absolute_scale[level_number],MAXIMA_NORMALIZATION);
	float step_advection = 1.f;
	
	BackwardSampling_CUDA(LOCAL_DOMAIN,local_color_field_array,advection_field_array,10,3,0.f,0.f,true,0.f,step_advection);
	cutilSafeCall( hipMemcpyToArray(hierarchical_color_field_array[level_number], 0, 0, float4_buffer, width*height*sizeof(float4), hipMemcpyDeviceToDevice ) );
}

////////////////////////////////////////////// GRADIENT ACCUMULATION
__global__ void AccumulateGradient_Kernel(float3 * dst_grad_w,float3 * dst_grad_h,const unsigned int width,const unsigned int height, const unsigned int iter_num,const unsigned int subdiv,const unsigned int it_w,const unsigned int it_h, const float step_amplification)
{

	int index_w = (blockIdx.x*subdiv + it_w)*BLOCK_SIZE_X + threadIdx.x;
 	int index_h = (blockIdx.y*subdiv + it_h)*BLOCK_SIZE_Y + threadIdx.y;

	if( (index_w) < width && (index_h) < height ){

        float dual_pos_w = (float)(index_w) + 0.5f;
		float dual_pos_h = (float)(index_h) + 0.5f;
		float4 center = tex2D(color_texture,dual_pos_w,dual_pos_h); 
		float2 advection_direction;

		// Advect grad_w
		if( index_w < width-1){
			
			float pos_w = dual_pos_w + 0.5f;
			float pos_h = dual_pos_h;

			for(int i =0; i<iter_num; i++){
				advection_direction = tex2D(advection_texture,pos_w,pos_h);
				pos_w += advection_direction.x*step_amplification;
				pos_h += advection_direction.y*step_amplification;
			}

			if((pos_w >= 1.f && pos_w <= (float)(width) - 1.f) && (pos_h >= 0.5f && pos_h <= (float)(height) -0.5f)){
				int cw = (int)floor(pos_w);
				int ch = (int)floor(pos_h-0.5f);
				float dw = (pos_w)-(float)(cw);
				float dh = (pos_h-0.5f)-(float)(ch);

				float4 neighbour = tex2D(color_texture,dual_pos_w + 1.f,dual_pos_h); 
				float3 grad = SubstractionFloat4ToFloat3(neighbour,center);
				for(int i=0; i<BLOCK_SIZE_X; i++){
					for(int j=0; j<BLOCK_SIZE_Y; j++){
					if(threadIdx.x == i && threadIdx.y ==j){

						dst_grad_w[cw+ch*(width+1)].x += grad.x*(1.f-dw)*(1.f-dh);
						dst_grad_w[cw+ch*(width+1)].y += grad.y*(1.f-dw)*(1.f-dh);
						dst_grad_w[cw+ch*(width+1)].z += grad.z*(1.f-dw)*(1.f-dh);

						dst_grad_w[cw+1+ch*(width+1)].x += grad.x*dw*(1.f-dh);
						dst_grad_w[cw+1+ch*(width+1)].y += grad.y*dw*(1.f-dh);
						dst_grad_w[cw+1+ch*(width+1)].z += grad.z*dw*(1.f-dh);

						dst_grad_w[cw + (ch+1)*(width+1)].x += grad.x*(1.f-dw)*dh;
						dst_grad_w[cw + (ch+1)*(width+1)].y += grad.y*(1.f-dw)*dh;
						dst_grad_w[cw + (ch+1)*(width+1)].z += grad.z*(1.f-dw)*dh;

						dst_grad_w[cw + 1 + (ch+1)*(width+1)].x += grad.x*dw*dh;
						dst_grad_w[cw + 1 + (ch+1)*(width+1)].y += grad.y*dw*dh;
						dst_grad_w[cw + 1 + (ch+1)*(width+1)].z += grad.z*dw*dh;
						}
					}
				}
			}
		}

		if(index_h < height-1){
		
			float pos_w = dual_pos_w;
			float pos_h = dual_pos_h + 0.5f;

			for(int i =0; i<iter_num; i++){
				advection_direction = tex2D(advection_texture,pos_w,pos_h);
				pos_w += advection_direction.x*step_amplification;
				pos_h += advection_direction.y*step_amplification;
			}

			if((pos_w >= 0.5 && pos_w <= (float)(width) -0.5f) && (pos_h >= 1.f && pos_h <= (float)(height) -1.f)){
				int cw = (int)floor(pos_w-0.5f);
				int ch = (int)floor(pos_h);
				float dw = (pos_w)-(float)(cw-0.5f);
				float dh = (pos_h)-(float)(ch);

				float4 neighbour = tex2D(color_texture,dual_pos_w,dual_pos_h + 1.f); 
				float3 grad = SubstractionFloat4ToFloat3(neighbour,center);
				for(int i=0; i<BLOCK_SIZE_X; i++){
					for(int j=0; j<BLOCK_SIZE_Y; j++){
					if(threadIdx.x == i && threadIdx.y ==j){
						dst_grad_h[cw+ch*(width)].x += grad.x*(1.f-dw)*(1.f-dh);
						dst_grad_h[cw+ch*(width)].y += grad.y*(1.f-dw)*(1.f-dh);
						dst_grad_h[cw+ch*(width)].z += grad.z*(1.f-dw)*(1.f-dh);

						dst_grad_h[cw+1+ch*(width)].x += grad.x*dw*(1.f-dh);
						dst_grad_h[cw+1+ch*(width)].y += grad.y*dw*(1.f-dh);
						dst_grad_h[cw+1+ch*(width)].z += grad.z*dw*(1.f-dh);


						dst_grad_h[cw + (ch+1)*(width)].x += grad.x*(1.f-dw)*dh;
						dst_grad_h[cw + (ch+1)*(width)].y += grad.y*(1.f-dw)*dh;
						dst_grad_h[cw + (ch+1)*(width)].z += grad.z*(1.f-dw)*dh;

						dst_grad_h[cw + 1 + (ch+1)*(width)].x += grad.x*dw*dh;
						dst_grad_h[cw + 1 + (ch+1)*(width)].y += grad.y*dw*dh;
						dst_grad_h[cw + 1 + (ch+1)*(width)].z += grad.z*dw*dh;
						}
					}
				}
			}
		}
	}
}

__global__ void Divergence_Kernel(float3 * src_w,float3 * src_h,float* dst, const unsigned int imgWidth,const unsigned int imgHeight, int channel)// COMPUTES MINUS DIVERGENCE
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;
	if( global_id_x < imgWidth && global_id_y < imgHeight){
			int write_pos = global_id_x + global_id_y*imgWidth;
			if(channel ==0)
				dst[write_pos] =  (src_w[write_pos+global_id_y].x - src_w[write_pos+global_id_y+1].x) + (src_h[write_pos].x - src_h[write_pos + imgWidth].x);
			else if(channel == 1)
				dst[write_pos] =  (src_w[write_pos+global_id_y].y - src_w[write_pos+global_id_y+1].y) + (src_h[write_pos].y - src_h[write_pos + imgWidth].y);
			else if(channel == 2)
				dst[write_pos] =  (src_w[write_pos+global_id_y].z - src_w[write_pos+global_id_y+1].z) + (src_h[write_pos].z - src_h[write_pos + imgWidth].z);
	}
}

void CUDA_Advection_Object::Divergence_CUDA(float3 * src_w,float3 * src_h,float* dst, const unsigned int imgWidth,const unsigned int imgHeight, int channel)// COMPUTES MINUS DIVERGENCE
{
	unsigned int  blocksW = (unsigned int) ceilf( (float) width / (float) BLOCK_SIZE );
    unsigned int  blocksH = (unsigned int) ceilf( (float) height /(float) BLOCK_SIZE );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );

	Divergence_Kernel<<< gridDim, blockDim >>>(gradient_accumulation_array_w,gradient_accumulation_array_h,float_buffer_0,width,height,channel);
}

void CUDA_Advection_Object::AccumulateGradient_CUDA(const unsigned int num_iter , const float step_amplification)
{

	cutilSafeCall(hipMemset(gradient_accumulation_array_w, 0, ( width+1 )* height * sizeof(float3)));
	cutilSafeCall(hipMemset(gradient_accumulation_array_h, 0, width * (height +1) * sizeof(float3)));

	color_texture.addressMode[0] = hipAddressModeMirror;
	color_texture.addressMode[1] = hipAddressModeMirror;
	color_texture.filterMode = hipFilterModeLinear;
	color_texture.normalized = false;

	cutilSafeCall( hipBindTextureToArray(color_texture,local_color_field_array) );

	advection_texture.addressMode[0] = hipAddressModeMirror;
	advection_texture.addressMode[1] = hipAddressModeMirror;
	advection_texture.filterMode = hipFilterModeLinear;
	advection_texture.normalized = false; // NOTE : This may changed if out of range coordinates are used!!

	cutilSafeCall( hipBindTextureToArray(advection_texture,advection_field_array) );

	const unsigned int subdiv =2;

	const unsigned int num_threads_w = ceilf( (float) width / (float) (subdiv) );
	const unsigned int num_threads_h = ceilf( (float) height / (float) (subdiv) );

	const unsigned int  blocksW = (unsigned int) ceilf( (float) num_threads_w / (float) BLOCK_SIZE_X );
    const unsigned int  blocksH = (unsigned int) ceilf( (float) num_threads_h /(float) BLOCK_SIZE_Y );
    dim3 gridDim( blocksW, blocksH, 1 );
    dim3 blockDim( BLOCK_SIZE_X, BLOCK_SIZE_Y, 1 );

	for(int it_w =0; it_w<subdiv; it_w++){
			for(int it_h =0; it_h<subdiv; it_h++){
				AccumulateGradient_Kernel<<< gridDim, blockDim >>>(gradient_accumulation_array_w,gradient_accumulation_array_h, width, height,num_iter,subdiv,it_w,it_h,step_amplification);
			}
	}
	cutilSafeCall( hipUnbindTexture(color_texture) );
    cutilSafeCall( hipUnbindTexture(advection_texture) );
		
}


__global__ void Gradient_Kernel(float3 * gradient_accumulation_array_w, float3 * gradient_accumulation_array_h, const int width, const int height)
{
	int global_id_x= blockIdx.x*BLOCK_SIZE + threadIdx.x;
	int global_id_y= blockIdx.y*BLOCK_SIZE + threadIdx.y;
	if( global_id_x < width  && global_id_y < height){

		float pos_w = (float)(global_id_x) + 0.5f;
		float pos_h = (float)(global_id_y) + 0.5f;

		int write_pos  = global_id_x + global_id_y*width;
		float4 center = tex2D(color_texture,pos_w,pos_h); 

		if( global_id_x < width-1){
		
		float4 neighbour_right = tex2D(color_texture,pos_w + 1.f,pos_h); 
		write_pos = (global_id_x + 1) + global_id_y*(width+1);

		gradient_accumulation_array_w[write_pos] = SubstractionFloat4ToFloat3(neighbour_right,center);
		}

		if(global_id_y < height-1){
		
		float4 neighbour_up = tex2D(color_texture,pos_w,pos_h + 1.f); 
		write_pos = global_id_x+ (global_id_y + 1)*width;
		gradient_accumulation_array_h[write_pos] = SubstractionFloat4ToFloat3(neighbour_up,center);
		}
	}
}

void CUDA_Advection_Object::SolveGradientField_CUDA()
{
	for(int i=0; i<3; i++){
		 CopyFloatFromFloat4(float4_buffer,float_buffer_0,width,height,i);
		 thrust::device_ptr<float> dev_ptr(float_buffer_0);
		 float dc = thrust::reduce(dev_ptr, dev_ptr + width * height, (float) 0.f ,thrust::plus<float>()); 
		 Divergence_CUDA(gradient_accumulation_array_w,gradient_accumulation_array_h,float_buffer_0,width,height,i);
		 GPUSolvers::FFTLaplaceSolver(float_buffer_0,fft_padding_buffer,fft_complex_buffer,fftPlanFwd, fftPlanInv,width,height,dc);
		 CopyFloatToFloat4(float_buffer_0,float4_buffer,width,height,i);
	}
}

